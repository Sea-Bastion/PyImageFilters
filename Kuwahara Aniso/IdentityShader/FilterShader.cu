 
#include <hip/hip_runtime.h>
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-functions

extern "C"
__global__ void Kuwahara(hipTextureObject_t Image, int *ImageShape, float *Mask, int MaskSize, unsigned char *OutImage){


//	int MaskRadius = MaskSize/2;

	int y = blockIdx.x;
	int x = threadIdx.x;
	
	int CenterIndex = y * ImageShape[1] + x;
	CenterIndex *= 4;

	uchar4 PixelData = tex2D<uchar4>(Image, x, y);

	printf ("%.3f\n ", (float)PixelData.x );
	OutImage[CenterIndex + 0] = PixelData.x;
	OutImage[CenterIndex + 1] = PixelData.y;
	OutImage[CenterIndex + 2] = PixelData.z;
	OutImage[CenterIndex + 3] = 1;


}


__global__ void BoxBlur(double *buffer, int filter_size, double *return_value){

	double sum = 0;

	for (int y = 0; y <filter_size ;y++){
		sum += buffer[y];
	}

	sum /= filter_size;
	return_value[0] = sum;


}
