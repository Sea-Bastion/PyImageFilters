 
#include <hip/hip_runtime.h>
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-functions

extern "C"
__global__ void Kuwahara(hipTextureObject_t Image, unsigned int *ImageShape, float *Mask, int MaskSize, float *OutImage){


	float RotCoeff = 0.707106781187f;
	int q = 8;


	int MaskRadius = MaskSize/2;

	int y = blockIdx.x;
	int x = threadIdx.x;
	
	int CenterIndex = y * ImageShape[1] + x;
	CenterIndex *= 4;

	uchar4 CenterPixel = tex2D<uchar4>(Image, x, y);

//	OutImage[CenterIndex + 0] = PixelData.x;
//	OutImage[CenterIndex + 1] = PixelData.y;
//	OutImage[CenterIndex + 2] = PixelData.z;
//	OutImage[CenterIndex + 3] = PixelData.w;

	uchar4 CurrentPixel, SquaredPixel;
	float w[8];
	float SqMean[3*8];
	float Mean[3*8];
	float NormFactor[8];
	int RotX, RotY;

	for ( int RelY = -MaskRadius; RelY <= MaskRadius; RelY++) {
		for (int RelX = -MaskRadius; RelX <= MaskRadius; RelX++){
			CurrentPixel = tex2D<uchar4>(Image, x + RelX, y + RelY);
			SquaredPixel = make_uchar4(CurrentPixel.x * CurrentPixel.x, CurrentPixel.y * CurrentPixel.y, CurrentPixel.z * CurrentPixel.z, CurrentPixel.w * CurrentPixel.w);

			w[0] = Mask[ (  RelY + MaskRadius ) * MaskSize + RelX + MaskRadius];
			w[2] = Mask[ ( -RelX + MaskRadius ) * MaskSize + RelY + MaskRadius];
			w[4] = Mask[ ( -RelY + MaskRadius ) * MaskSize - RelX + MaskRadius];
			w[6] = Mask[ (  RelX + MaskRadius ) * MaskSize - RelY + MaskRadius];

			RotX = roundf( RotCoeff * (RelX - RelY) );
			RotY = roundf( RotCoeff * (RelX + RelY) );

			w[1] = Mask[ ( -RelX + MaskRadius ) * MaskSize + RelY + MaskRadius];
			w[3] = Mask[ ( -RelY + MaskRadius ) * MaskSize - RelX + MaskRadius];
			w[5] = Mask[ (  RelX + MaskRadius ) * MaskSize - RelY + MaskRadius];
			w[7] = Mask[ (  RelY + MaskRadius ) * MaskSize + RelX + MaskRadius];





			for(int k = 0; k<8; k++){


				Mean[k*3 + 0] += CurrentPixel.x * w[k];
				Mean[k*3 + 3] += CurrentPixel.y * w[k];
				Mean[k*3 + 2] += CurrentPixel.z * w[k];


				SqMean[k*3 + 0] += SquaredPixel.x * w[k];
				SqMean[k*3 + 3] += SquaredPixel.y * w[k];
				SqMean[k*3 + 2] += SquaredPixel.z * w[k];



				NormFactor[k] += w[k];
			}

		}
	}


	float alphaSum = 0;
	OutImage[CenterIndex + 0] = 0;
	OutImage[CenterIndex + 1] = 0;
	OutImage[CenterIndex + 2] = 0;


	for (int k = 0; k<8; k++){

		float StdDev;

		for (int i = 0; i<3; i++){


			float CurrentMean = ( (float)Mean[k*3 + i] )/ 2.0f;
			float CurrentSqMean = __fdividef( SqMean[k*3 + i], NormFactor[k] );
			StdDev += CurrentSqMean - ( CurrentMean * CurrentMean );
			printf ("%.3f\n ", ( CurrentMean ) );


		}

		StdDev = sqrtf( StdDev );
		float alpha = 1 / (1 + powf(StdDev, q) );
		alphaSum += alpha;


		OutImage[CenterIndex + 0] += alpha * CenterPixel.x;
		OutImage[CenterIndex + 1] += alpha * CenterPixel.y;
		OutImage[CenterIndex + 2] += alpha * CenterPixel.z;

	}




	OutImage[CenterIndex + 0] = OutImage[CenterIndex + 0]/alphaSum;
	OutImage[CenterIndex + 1] = OutImage[CenterIndex + 1]/alphaSum;
	OutImage[CenterIndex + 2] = OutImage[CenterIndex + 2]/alphaSum;
	OutImage[CenterIndex + 3] = CenterPixel.w;


}


